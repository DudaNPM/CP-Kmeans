#include "hip/hip_runtime.h"
#include "../include/k_means_cuda.h"



// Final print

__host__ void print_info(int iterations)  {
    printf("N = %d, K = %d\n", NUM_SAMPLES, NUM_CLUSTERS);
    for (register int k = 0; k < NUM_CLUSTERS; k++)
        printf("Center: (%.3f, %.3f) : Size: %d\n", h_clusters[k].pos.x, h_clusters[k].pos.y, h_clusters[k].size);
    printf("Iterations: %d\n", iterations);
}



// Initialization: N points and K clusters

__host__ void initialize() {
    h_points   = (Point *)   malloc(NUM_SAMPLES * sizeof(Point)  );
    h_clusters = (Cluster *) malloc(NUM_CLUSTERS * sizeof(Cluster));

    h_sum_x = (float *) malloc(NUM_CLUSTERS * sizeof(float));
    h_sum_y = (float *) malloc(NUM_CLUSTERS * sizeof(float));
    h_size  = (int *)   malloc(NUM_CLUSTERS * sizeof(int)  );

    srand(10);
    for(register int i = 0; i < NUM_SAMPLES; i++) {
        h_points[i].x = (float) rand() / RAND_MAX;
        h_points[i].y = (float) rand() / RAND_MAX;
    }
    
    for(register int k = 0; k < NUM_CLUSTERS; k++) {
        h_clusters[k].pos.x = h_points[k].x;
        h_clusters[k].pos.y = h_points[k].y;
    }
}



// Euclidean distance between 2 points
// (no need to use sqrt, we dont need the right dist)

__device__ float dist(Point a, Point b) {
  return (a.x-b.x)*(a.x-b.x) + (a.y-b.y)*(a.y-b.y);
}



__global__ void k_means_it_kernel(Point * d_points, Cluster * d_clusters, float * d_sum_x, float * d_sum_y, int * d_size) {
    register int i = blockIdx.x * blockDim.x + threadIdx.x;  // global thread id
    if (i >= NUM_SAMPLES) return;                            // in case there are more threads than points

    register int cluster_id = 0;
    register float lowest_dist = dist(d_points[i],d_clusters[0].pos);

    for (register int k = 1; k < NUM_CLUSTERS; k++) {
        register float distance = dist(d_points[i],d_clusters[k].pos);
        if (distance < lowest_dist) {
            lowest_dist = distance;
            cluster_id = k;
        }
    }

    atomicAdd(&d_size[cluster_id] , 1            );
    atomicAdd(&d_sum_x[cluster_id], d_points[i].x);
    atomicAdd(&d_sum_y[cluster_id], d_points[i].y);
}



__host__ void k_means_it(Point * d_points, Cluster * d_clusters, float * d_sum_x, float * d_sum_y, int * d_size) {
    // Iniatilize/clear device variabels to 0
    hipMemset(d_size , 0, NUM_CLUSTERS * sizeof(int)  );
    hipMemset(d_sum_x, 0, NUM_CLUSTERS * sizeof(float));
    hipMemset(d_sum_y, 0, NUM_CLUSTERS * sizeof(float));

    // Copy memory to the device
    hipMemcpy(d_clusters, h_clusters, NUM_CLUSTERS * sizeof(Cluster), hipMemcpyHostToDevice);

    // Launch the kernel
    k_means_it_kernel<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_points, d_clusters, d_sum_x, d_sum_y, d_size);

    // Copy memory to the host
    hipMemcpy(h_size , d_size , NUM_CLUSTERS * sizeof(int)  , hipMemcpyDeviceToHost);
    hipMemcpy(h_sum_x, d_sum_x, NUM_CLUSTERS * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_sum_y, d_sum_y, NUM_CLUSTERS * sizeof(float), hipMemcpyDeviceToHost);

    // Calculate centroids and size of the clusters
    for (register int k = 0; k < NUM_CLUSTERS; k++) {
        h_clusters[k].pos.x = h_sum_x[k] / h_size[k];
        h_clusters[k].pos.y = h_sum_y[k] / h_size[k];
        h_clusters[k].size  = h_size[k];
    }
}



// Main

__host__ int main(int argc, char *argv[]) {
    initialize();                   // initialize samples and clusters
    register int iterations = -1;   // initial iteration is inside the while

    // Device variables
    int * d_size;           // array of size K with clusters size
    float * d_sum_x;        // array of size K with sum of all x of each cluster
    float * d_sum_y;        // array of size K with sum of all y of each cluster
    Point * d_points;       // N samples
    Cluster * d_clusters;   // K clusters

    // Allocate memory on the device
    hipMalloc((void**) &d_size    , NUM_CLUSTERS * sizeof(int)    );
    hipMalloc((void**) &d_sum_x   , NUM_CLUSTERS * sizeof(float)  );
    hipMalloc((void**) &d_sum_y   , NUM_CLUSTERS * sizeof(float)  );
    hipMalloc((void**) &d_points  , NUM_SAMPLES  * sizeof(Point)  );
    hipMalloc((void**) &d_clusters, NUM_CLUSTERS * sizeof(Cluster));

    // Copy memory to the device
    hipMemcpy(d_points, h_points, NUM_SAMPLES * sizeof(Point), hipMemcpyHostToDevice);

    while (iterations < 20) {
        k_means_it(d_points,d_clusters,d_sum_x,d_sum_y,d_size);
        iterations++;
    }

    // Print info
    print_info(iterations);

    // Free the device memory
    hipFree(d_sum_x);
    hipFree(d_sum_y);
    hipFree(d_size);
    hipFree(d_points);
    hipFree(d_clusters);

    // Free the host memory
    free(h_points);
    free(h_clusters);
    free(h_sum_x);
    free(h_sum_y);
    free(h_size);

    return 0;
}
